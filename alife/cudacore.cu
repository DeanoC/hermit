#include "al2o3_platform/platform.h"
#include "al2o3_memory/memory.h"
#include <hip/hip_runtime.h>

inline int _ConvertSMVer2Cores(int major, int minor) {
	// Defines for GPU Architecture types (using the SM version to determine
	// the # of cores per SM
	typedef struct {
		int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
		// and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] = {
			{0x30, 192},
			{0x32, 192},
			{0x35, 192},
			{0x37, 192},
			{0x50, 128},
			{0x52, 128},
			{0x53, 128},
			{0x60, 64},
			{0x61, 128},
			{0x62, 128},
			{0x70, 64},
			{0x72, 64},
			{0x75, 64},
			{-1, -1}};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1) {
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// If we don't find the values, we default use the previous one
	// to run properly
	LOGINFO("MapSMtoCores for SM %d.%d is undefined. Default to use %d Cores/SM\n",
					major, minor, nGpuArchCoresPerSM[index - 1].Cores);
	return nGpuArchCoresPerSM[index - 1].Cores;
}

inline const char *_ConvertSMVer2ArchName(int major, int minor) {
	// Defines for GPU Architecture types (using the SM version to determine
	// the GPU Arch name)
	typedef struct {
		int SM;  // 0xMm (hexidecimal notation), M = SM Major version,
		// and m = SM minor version
		const char *name;
	} sSMtoArchName;

	sSMtoArchName nGpuArchNameSM[] = {
			{0x30, "Kepler"},
			{0x32, "Kepler"},
			{0x35, "Kepler"},
			{0x37, "Kepler"},
			{0x50, "Maxwell"},
			{0x52, "Maxwell"},
			{0x53, "Maxwell"},
			{0x60, "Pascal"},
			{0x61, "Pascal"},
			{0x62, "Pascal"},
			{0x70, "Volta"},
			{0x72, "Xavier"},
			{0x75, "Turing"},
			{-1, "Graphics Device"}};

	int index = 0;

	while (nGpuArchNameSM[index].SM != -1) {
		if (nGpuArchNameSM[index].SM == ((major << 4) + minor)) {
			return nGpuArchNameSM[index].name;
		}

		index++;
	}

	// If we don't find the values, we default use the previous one
	// to run properly
	LOGINFO("MapSMtoArchName for SM %d.%d is undefined. Default to use %s\n",
					major,
					minor,
					nGpuArchNameSM[index - 1].name);
	return nGpuArchNameSM[index - 1].name;
}

template<typename T>
void check(T result, char const *const func, const char *const file,
					 int const line) {
	if (result) {
		LOGERROR("CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line,
						 static_cast<unsigned int>(result), hipGetErrorName(result), func);
	}
}

// This will output the proper CUDA error strings in the event
// that a CUDA host call returns an error
#define checkCudaErrors(val) check((val), #val, __FILE__, __LINE__)

struct Cuda {
	int deviceIndex;
};

Cuda *CUDACore_Create() {

	int deviceCount;
	int pickedDeviceIndex = -1;
	int pickedTotalCores = -1;
	hipDeviceProp_t pickedDevice{};
	checkCudaErrors(hipGetDeviceCount(&deviceCount));
	LOGINFO("--- CUDA Devices ---");

	for (int i = 0u; i < deviceCount; ++i) {
		hipDeviceProp_t deviceProp;
		int computeMode = -1;
		checkCudaErrors(hipDeviceGetAttribute(&computeMode, hipDeviceAttributeComputeMode, i));

		if(computeMode == hipComputeModeProhibited) continue;

		checkCudaErrors(hipGetDeviceProperties(&deviceProp, i));

		int const coresPerSM = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
		int const totalCores = coresPerSM * deviceProp.multiProcessorCount;
		int const computePerf = totalCores * (deviceProp.clockRate/1024);

		LOGINFO("%d: %s %s (%d.%d)", i,
				deviceProp.name, _ConvertSMVer2ArchName(deviceProp.major, deviceProp.minor), deviceProp.major, deviceProp.minor);
		LOGINFO("%d: SMs %d, Cores %d, Total Cores %d Clock %d ~GFLOPs %f", i,
				deviceProp.multiProcessorCount, coresPerSM, totalCores, deviceProp.clockRate/1024, ((float)2 * computePerf)/1024.0f);

		// for now just pick the biggest new enough device
		if (totalCores > pickedTotalCores) {
			memcpy(&pickedDevice, &deviceProp, sizeof(hipDeviceProp_t));
			pickedDeviceIndex = i;
			pickedTotalCores = totalCores;
		}
	}

	LOGINFO("---");

	if (pickedDeviceIndex == -1) {
		return nullptr;
	}

	checkCudaErrors(hipSetDevice(pickedDeviceIndex));

	Cuda* cuda = (Cuda*)MEMORY_CALLOC(1, sizeof(Cuda));
	if(!cuda) return nullptr;

	cuda->deviceIndex = pickedDeviceIndex;
	return cuda;
}

void CUDACore_Destroy(Cuda *cuda) {
	if(!cuda) return;

	MEMORY_FREE(cuda);
}